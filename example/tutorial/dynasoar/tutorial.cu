#include "hip/hip_runtime.h"
#include "tutorial.h"
#include "dynasoar.h"

__device__ AllocatorT* device_allocator;        // device side
AllocatorHandle<AllocatorT>* allocator_handle;  // host side

int DEFAULT = 36;

int main(int argc, char** argv)
{
  int n = DEFAULT;
  if (argc == 2) {
    n = atoi(argv[1]);
    if (n == 0) n = DEFAULT;
  }

  // Some boilerplate code.... Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
    hipMemcpyHostToDevice);

  int h_result;
  int* d_result;

  hipMalloc(&d_result, sizeof(int));

  do_calc << <1, 1 >> > (n, d_result);
  hipDeviceSynchronize();

  struct timespec cpu_time_start, cpu_time_end;
  double cpu_time;
  timespec_get(&cpu_time_start, TIME_UTC);
  for (int i = 1; i < 200; i++)
  {
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (h_result != -1) {
      printf("-- Result: Fib(%i) = %i --\n", n, h_result);
      break;
    }
    // printf("====== Iteration: %i ======\n", i);
    allocator_handle->parallel_do<Fib, &Fib::calc>();
    // allocator_handle->parallel_do<Fib, &Fib::printInfo>();
    allocator_handle->parallel_do<Sum, &Sum::calc>();
    // allocator_handle->parallel_do<Sum, &Sum::calc>();
    // allocator_handle->parallel_do<Sum, &Sum::printInfo>();
    // allocator_handle->DBG_print_state_stats();
  }
  timespec_get(&cpu_time_end, TIME_UTC);
  cpu_time = (cpu_time_end.tv_sec - cpu_time_start.tv_sec) +
    (cpu_time_end.tv_nsec - cpu_time_start.tv_nsec) / 1e9;
  printf("fib_single(%d) = %d (%f sec)\n", n, h_result, cpu_time);
}

__global__ void do_calc(int n, int* result)
{
  *result = -1;
  new(device_allocator) Fib(result, n);
}

__device__ void Fib::calc()
{
  if (n <= 1) {
    *result = n;
    destroy(device_allocator, this);
    return;
  }
  Sum* sum = new(device_allocator) Sum(result);
  new(device_allocator) Fib(&sum->x, n - 1);
  result = &sum->y;
  n = n - 2;
  // new(device_allocator) Fib(&sum->y, n - 2);
  // destroy(device_allocator, this);
}

#ifdef PRINT_INFO
__device__ void Fib::printInfo()
{
  printf("N: %i\n", (int)n);
}
#endif

__device__ void Sum::calc()
{
  if (x != -1 && y != -1)
  {
    *result = x + y;
    destroy(device_allocator, this);
  }
}

#ifdef PRINT_INFO
__device__ void Sum::printInfo()
{
  printf("X: %i, Y: %i\n", (int)x, (int)y);
}
#endif
