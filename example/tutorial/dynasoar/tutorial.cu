#include "hip/hip_runtime.h"
#include "tutorial.h"
#include "dynasoar.h"

__device__ AllocatorT* device_allocator;        // device side
AllocatorHandle<AllocatorT>* allocator_handle;  // host side

class Bar : public AllocatorT::Base
{
public:
  declare_field_types(Bar, Foo*, int, int)

private:
  Field<Bar, 0> the_first_field_;
  Field<Bar, 1> the_second_field_;
  Field<Bar, 2> the_third_field_;

public:
  __device__ Bar(int a, int b)
    : the_first_field_(nullptr), the_second_field_(a), the_third_field_(b) {}

  __device__ void increment_by_one()
  {
    the_second_field_ += 1;
  }

  __device__ void increment_by_n(int n)
  {
    the_second_field_ += n;
  }

  __device__ void print_second()
  {
    printf("Second value: %i\n", (int)the_second_field_);
  }
};


class Foo : public AllocatorT::Base {
public:
  declare_field_types(Foo, int, int, int)

private:
  Field<Bar, 0> f0_;
  Field<Bar, 1> f1_;
  Field<Bar, 2> f2_;
};


__global__ void create_objs()
{
  Bar* result = new(device_allocator) Bar(threadIdx.x, 5);
}


int main(int argc, char** argv)
{
  // Some boilerplate code.... Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
    hipMemcpyHostToDevice);

  // Allocate a few objects.
  create_objs << <5, 10 >> > ();
  hipDeviceSynchronize();

  // Run a do-all operations in parallel.
  allocator_handle->parallel_do<Bar, &Bar::increment_by_one>();

  // If a member function takes an argument, we have to specify its type here.
  allocator_handle->parallel_do<Bar, int, &Bar::increment_by_n>(/*n=*/ 10);

  // Now print some stuff.
  allocator_handle->parallel_do<Bar, &Bar::print_second>();

  int result;
  bool isCalculated = false;

  do_calc << <1, 1 >> > (4, &result, &isCalculated);
  hipDeviceSynchronize();

  for (int i = 1; i < 7; i++)
  {
    if (isCalculated) {
      printf("-- Result: %i --\n", result);
      break;
    }
    printf("====== Iteration: %i ======\n", i);
    allocator_handle->parallel_do<Fib, &Fib::calc>();
    allocator_handle->parallel_do<Fib, &Fib::printInfo>();
    allocator_handle->parallel_do<Sum, &Sum::calc>();
    allocator_handle->parallel_do<Sum, &Sum::printInfo>();
  }
}

__global__ void do_calc(int n, int* result, bool* isCalculated)
{
  new(device_allocator) Fib(result, isCalculated, n);
}

__device__ void Fib::calc()
{
  if (n <= 1) {
    if (result) *result = n;
    if (isCalculated) *isCalculated = true;
    destroy(device_allocator, this);
    return;
  }
  Sum* sum = new(device_allocator) Sum(result, isCalculated);
  new(device_allocator) Fib(&sum->x, &sum->isXCalculated, n - 1);
  new(device_allocator) Fib(&sum->y, &sum->isYCalculated, n - 2);
  destroy(device_allocator, this);
}

__device__ void Fib::printInfo()
{
  printf("N: %i\n", (int)n);
}

__device__ void Sum::calc()
{
  if (isXCalculated && isYCalculated)
  {
    *result = x + y;
    *isCalculated = true;
    destroy(device_allocator, this);
  }
}

__device__ void Sum::printInfo()
{
  if (isXCalculated && isYCalculated)
  {
    printf("X: %i, Y: %i\n", (int)x, (int)y);
  }
  else if (isXCalculated)
  {
    printf("X: %i\n", (int)x);
  }
  else if (isYCalculated)
  {
    printf("Y: %i\n", (int)y);
  }
}
