#include "hip/hip_runtime.h"
#include "tutorial.h"
#include "dynasoar.h"

__device__ AllocatorT* device_allocator;        // device side
AllocatorHandle<AllocatorT>* allocator_handle;  // host side

int main(int argc, char** argv)
{
  // Some boilerplate code.... Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
    hipMemcpyHostToDevice);

  int h_result;
  int* d_result;

  hipMalloc(&d_result, sizeof(int));
  int n = 25;

  do_calc << <1, 1 >> > (n, d_result);
  hipDeviceSynchronize();

  for (int i = 1; i < 50; i++)
  {
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    if (h_result != -1) {
      printf("-- Result: Fib(%i) = %i --\n", n, h_result);
      break;
    }
    printf("====== Iteration: %i ======\n", i);
    allocator_handle->parallel_do<Fib, &Fib::calc>();
    // allocator_handle->parallel_do<Fib, &Fib::printInfo>();
    allocator_handle->parallel_do<Sum, &Sum::calc>();
    // allocator_handle->parallel_do<Sum, &Sum::printInfo>();
  }
}

__global__ void do_calc(int n, int* result)
{
  *result = -1;
  new(device_allocator) Fib(result, n);
}

__device__ void Fib::calc()
{
  if (n <= 1) {
    *result = n;
    destroy(device_allocator, this);
    return;
  }
  Sum* sum = new(device_allocator) Sum(result);
  new(device_allocator) Fib(&sum->x, n - 1);
  new(device_allocator) Fib(&sum->y, n - 2);
  destroy(device_allocator, this);
}

__device__ void Fib::printInfo()
{
  printf("N: %i\n", (int)n);
}

__device__ void Sum::calc()
{
  if (x != -1 && y != -1)
  {
    *result = x + y;
    destroy(device_allocator, this);
  }
}

__device__ void Sum::printInfo()
{
  printf("X: %i, Y: %i\n", (int)x, (int)y);
}
